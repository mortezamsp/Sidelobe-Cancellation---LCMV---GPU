#include "hip/hip_runtime.h"

using namespace std;
#include "matrix.h"


void Swap(double& a, double& b)
{
  double temp = a;
  a = b;
  b = temp;
}

matrix::matrix()
  {
    //printf("Executing constructor matrix() ...\n");
    // create a matrix object without content
    p = NULL;
    rows = 0;
    cols = 0;
  }
matrix::matrix(const int row_count, const int column_count)
  {
    // create a matrix object with given number of rows and columns
    p = NULL;

    if (row_count > 0 && column_count > 0)
    {
      rows = row_count;
      cols = column_count;

      p = new double*[rows];
      for (int r = 0; r < rows; r++)
      {
        p[r] = new double[cols];

        // initially fill in zeros for all values in the matrix;
        for (int c = 0; c < cols; c++)
        {
          p[r][c] = 0;
        }
      }
	
	  matrixSpaceHasBeenAllocated = true;
    }
  }
matrix::matrix(const int row_count, const int column_count, const char *fillingtype, double value)
  {
    // create a matrix object with given number of rows and columns
    p = NULL;
	srand(rows);
    if (row_count > 0 && column_count > 0)
    {
      rows = row_count;
      cols = column_count;

      p = new double*[rows];
      for (int r = 0; r < rows; r++)
      {
        p[r] = new double[cols];

        // initially fill in zeros for all values in the matrix;
        for (int c = 0; c < cols; c++)
        {
			if(strcmp(fillingtype,"rand") == 0)
			{
				p[r][c] = rand() / RAND_MAX;
			}
			else if(strcmp(fillingtype,"num") == 0)
			{
				p[r][c] = rand() / value;
			}
			else
			{
				p[r][c] = 0;
			}
        }
      }
    }
	matrixSpaceHasBeenAllocated = true;
  }

matrix::matrix(const int row_count, const int column_count, double *vector)
{
	p = NULL;
	if(row_count > 0 && column_count > 0)
	{
		rows = row_count;
		cols = column_count;

		p = new double* [rows];
		for(int i=0; i<rows; i++)
		{
			p[i] = new double[cols];
			for(int j=0; j<cols; j++)
				p[i][j] = vector[i*rows + j];
		}
		matrixSpaceHasBeenAllocated = true;
	}
}
matrix::matrix(const matrix& a)
  {
    rows = a.rows;
    cols = a.cols;
    p = new double*[a.rows];
    for (int r = 0; r < a.rows; r++)
    {
      p[r] = new double[a.cols];

      // copy the values from the matrix a
      for (int c = 0; c < a.cols; c++)
      {
        p[r][c] = a.p[r][c];
      }
    }
	matrixSpaceHasBeenAllocated = true;
  }

double matrix::sum()
  {
	  double s = 0;
	  for(int i=0; i<rows; i++)
		  for(int j=0; j<cols; j++)
			  s += p[i][j];
	  return s;
  }
double& matrix::operator()(const int r, const int c)
  {
    if (p != NULL && r >= 0 && r < rows && c >= 0 && c < cols)
    {
      return p[r][c];
    }
    else
    {
      throw MatrixException("Subscript out of range");
    }
  }
double matrix::get(const int r, const int c) const
  {
    if (p != NULL && r >= 0 && r < rows && c >= 0 && c < cols)
    {
      return p[r][c];
    }
    else
    {
      throw MatrixException("Subscript out of range");
    }
  }
matrix& matrix::operator= (const matrix& a)
  {
    rows = a.rows;
    cols = a.cols;
	delete []p;
    p = new double*[a.rows];
    for (int r = 0; r < a.rows; r++)
    {
      p[r] = new double[a.cols];

      // copy the values from the matrix a
      for (int c = 0; c < a.cols; c++)
      {
        p[r][c] = a.p[r][c];
      }
    }
	matrixSpaceHasBeenAllocated = true;
    return *this;
  }
matrix& matrix::Add(const double v)
  {
    for (int r = 0; r < rows; r++)
    {
      for (int c = 0; c < cols; c++)
      {
        p[r][c] += v;
      }
    }
     return *this;
  }
matrix& matrix::Subtract(const double v)
  {
    return Add(-v);
  }
matrix& matrix::Multiply(const double v)
{
	int memsize = sizeof(double)*rows*cols;
	double *a, *b;
	hipMalloc(&a, memsize);
	hipMemcpy(&a, (double *)&p, memsize, hipMemcpyHostToDevice);
	hipMalloc(&b, memsize);

	dim3 block(1, 256);
	matrixMultWithNumber<<<1, block>>>(a, b, v, memsize);
	
	hipMemcpy((double *)&p, &b, memsize, hipMemcpyHostToDevice);
	
	return *this;
}
matrix& matrix::Divide(const double v)
  {
     return Multiply(1/v);
  }
matrix matrix::Minor(const int row, const int col) const
  {
    matrix res;
    if (row >= 0 && row < rows && col >= 0 && col < cols)
    {
      res = matrix(rows, cols);

      // copy the content of the matrix to the minor, except the selected
      for (int r = 1; r <= (rows - (row >= rows)); r++)
      {
        for (int c = 1; c <= (cols - (col >= cols)); c++)
        {
          res(r - (r > row), c - (c > col)) = p[r-1][c-1];
        }
      }
    }
    else
    {
      throw MatrixException("Index for minor out of range");
    }

    return res;
  }
int matrix::Size(const int i) const
  {
    if (i == 1)
    {
      return rows;
    }
    else if (i == 2)
    {
      return cols;
    }
    return 0;
  }
int matrix::GetRows() const
  {
    return rows;
  }
int matrix::GetCols() const
  {
    return cols;
  }
matrix::~matrix()
  {
    delete p;
    p = NULL;
  }
matrix matrix::Inv()
{
  matrix res;
  double d = 0;    // value of the determinant

  d = Det();
  if (rows == cols && d != 0)
  {
    // this is a square matrix
    if (rows == 1)
    {
      // this is a 1 x 1 matrix
      res = matrix(rows, cols);
      res(0, 0) = 1 / get(1, 1);
    }
    else if (rows == 2)
    {
      // this is a 2 x 2 matrix
      res = matrix(rows, cols);
      res(0, 0) = get(2, 2);
      res(0, 1) = -get(1, 2);
      res(1, 0) = -get(2, 1);
      res(1, 1) = get(1, 1);
      res = (1/d) * res;
    }
    else
    {
      // this is a matrix of 3 x 3 or larger
      // calculate inverse using gauss-jordan elimination
      //   http://mathworld.wolfram.com/matrixInverse.html
      //   http://math.uww.edu/~mcfarlat/inverse.htm
      res = Diag(rows);   // a diagonal matrix with ones at the diagonal
	  matrix ai = *this;    // make a copy of matrix a

      for (int c = 0; c < cols; c++)
      {
        // element (c, c) should be non zero. if not, swap content
        // of lower rows
        int r;
        for (r = c; r < rows && ai(r, c) == 0; r++)
        {
        }
        if (r != c)
        {
          // swap rows
          for (int s = 0; s < cols; s++)
          {
            Swap(ai(c, s), ai(r, s));
            Swap(res(c, s), res(r, s));
          }
        }

        // eliminate non-zero values on the other rows at column c
        for (int r = 0; r < rows; r++)
        {
          if(r != c)
          {
            // eleminate value at column c and row r
            if (ai(r, c) != 0)
            {
              double f = - ai(r, c) / ai(c, c);

              // add (f * row c) to row r to eleminate the value
              // at column c
              for (int s = 0; s < cols; s++)
              {
                ai(r, s) += f * ai(c, s);
                res(r, s) += f * res(c, s);
              }
            }
          }
          else
          {
            // make value at (c, c) one,
            // divide each value on row r with the value at ai(c,c)
            double f = ai(c, c);
            for (int s = 0; s < cols; s++)
            {
              ai(r, s) /= f;
              res(r, s) /= f;
            }
          }
        }
      }
    }
  }
  else
  {
    if (rows == cols)
    {
      //throw MatrixException("matrix must be square");
	#ifdef win32
		return matrix(cols, rows, "num", LONG_MAX);
	#else
		return matrix(cols, rows, "num", 99999999);
	#endif
    }
    else
    {
      throw MatrixException("Determinant of matrix is zero");
    }
  }
  return res;
}
#include<sys/time.h>
matrix matrix::transpose()
{
	matrix res(cols, rows);
	for(int i=0; i<rows; i++)
		for(int j=0; j<cols; j++)
			res(j, i) = get(i, j);
	return res;
}
double matrix::Det()
{
	//using sarrus rule :

  double sum1 = 0;
  for(int j=0; j<cols; j++)
  {
	  double ts=1;
	  for(int i=0; i<rows; i++)
		  ts *= p[i][(i+j) % cols];
	  sum1 += ts;
  }

  double sum2 = 0;
  for(int j=cols-1; j>0; j--)
  {
	  double ts=1;
	  for(int i=0; i<rows; i++)
		  ts *= p[i][(j-i+rows-1) % cols];
	  sum2 += ts;
  }

  return sum1 - sum2;
}
matrix matrix::Diag(const int n)
{
  matrix res = matrix(n, n);
  for (int i = 0; i < n; i++)
  {
    res(i, i) = 1;
  }
  return res;
}
matrix matrix::Diag()
{
  matrix res;
  if (cols == 1)
  {
    // the given matrix is a vector n x 1
    res = matrix(rows, rows);

    // copy the values of the vector to the matrix
    for (int r=1; r <= rows; r++)
    {
      res(r, r) = get(r, 1);
    }
  }
  else if (rows == 1)
  {
    // the given matrix is a vector 1 x n
    res = matrix(cols, cols);

    // copy the values of the vector to the matrix
    for (int c=1; c <= cols; c++)
    {
      res(c, c) = get(1, c);
    }
  }
  else
  {
    throw MatrixException("Parameter for diag must be a vector");
  }
  return res;
}

matrix:: matrix operator * (const matrix& a, const matrix& b)
{
                // check if the dimensions match
		int ar = a.GetRows(),
		    ac = a.GetCols(),
		    br = b.GetRows(),
		    bc = b.GetCols();
                if (ac == br)
                {
                        double *c_, *a_, *b_;
                        hipMalloc(&c_, sizeof(double)*ar*bc);
                        hipMalloc(&a_, sizeof(double)*ar*ac);
                        hipMalloc(&b_, sizeof(double)*br*bc);
			double *av = a.GetVector(),
			       *bv = b.GetVector();
                        hipMemcpy(&a_, (double *)&av, sizeof(double)*ar*ac, hipMemcpyHostToDevice);
                        hipMemcpy(&b_, (double *)&bv, sizeof(double)*br*bc, hipMemcpyHostToDevice);

                        dim3 block(block_size, block_size);
                        dim3 grid(floor(bc/block_size), floor(ar/block_size));
                        matrixMul<<<grid, block>>>(c_, a_, b_, ac, bc);

                        double *c = new double[ar*bc];
                        hipMemcpy(&c, &c_, sizeof(double)*ar*bc, hipMemcpyDeviceToHost);
                        matrix res(ar, bc, c);

			hipFree(c_);
			hipFree(a_);
			hipFree(b_);

                        return res;
                }
                else
                        throw MatrixException("\nERRORE : niga kardi bebini dari chikar mikoni? andeze ye matris ha barabar nist!");
}

matrix:: matrix operator + (const matrix& a, const matrix& b)
{
                // check if the dimensions match
                int ar = a.GetRows(),
                    ac = a.GetCols(),
                    br = b.GetRows(),
                    bc = b.GetCols();
                if (ar==br && ac==bc)
		{
			double *a_ , *b_;
			int memsize = a.GetCols() * a.GetRows();
			hipMalloc(&a_, memsize*sizeof(double));
			hipMemcpy(&a_, a.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);
			hipMalloc(&b_, memsize*sizeof(double));
			hipMemcpy(&b_, b.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);

			dim3 block(1, 256);
			MatrixAdd<<<1, block>>>(a_, b_, memsize);

			double *c = new double[memsize];
			hipMemcpy(&c, &a_, memsize*sizeof(double), hipMemcpyDeviceToHost);
			matrix res(a.GetRows(), a.GetCols(), c);
			
			hipFree(c);
			hipFree(a_);
			hipFree(b_);
		
			return res;
                }
                else
                	throw MatrixException("Dimensions does not match");
}/*
matrix:: matrix operator += (const matrix& a, const matrix& b)
{
		// check if the dimensions match
                int ar = a.GetRows(),
                    ac = a.GetCols(),
                    br = b.GetRows(),
                    bc = b.GetCols();
                if (ar==br && ac==bc)
		{
                        double *a_ , *b_;
                        int memsize = a.GetCols() * a.GetRows();
                        hipMalloc(&a_, memsize*sizeof(double));
                        hipMemcpy(&a_, a.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);
                        hipMalloc(&b_, memsize*sizeof(double));
                        hipMemcpy(&b_, b.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);
			
                        dim3 block(1, 256);
                        MatrixAdd<<<1, block>>>(a_, b_, memsize);
			
			double *c = new double[memsize];
                        hipMemcpy(&c, &a_, memsize*sizeof(double), hipMemcpyDeviceToHost);
			matrix res(a.GetRows(), a.GetCols(), c);
			
			hipFree(c);
                        hipFree(a_);
                        hipFree(b_);
			
			return res;
                }
                else
                        throw MatrixException("Dimensions does not match");
}
matrix:: matrix  operator+ (const matrix& a, const double b)
{
                        double *a_;
                        int memsize = a.GetCols() * a.GetRows();
                        hipMalloc(&a_, memsize*sizeof(double));
                        hipMemcpy(&a_, a.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);

                        dim3 block(1, 256);
                        MatrixAddWithNumber<<<1, block>>>(a_, b, memsize);

                        double *c = new double[memsize];
                        hipMemcpy(&c, &a_, memsize*sizeof(double), hipMemcpyDeviceToHost);
                        matrix res(a.GetRows(), a.GetCols(), c);

                        hipFree(c);
                        hipFree(a_);

                        return res;
}
matrix:: matrix operator+ (const double b, const matrix& a)
{
                        double *a_;
                        int memsize = a.GetCols() * a.GetRows();
                        hipMalloc(&a_, memsize*sizeof(double));
                        hipMemcpy(&a_, a.GetVector(), memsize*sizeof(double), hipMemcpyHostToDevice);

                        dim3 block(1, 256);
                        MatrixAddWithNumber<<<1, block>>>(a_, b, memsize);

                        double *c = new double[memsize];
                        hipMemcpy(&c, &a_, memsize*sizeof(double), hipMemcpyDeviceToHost);
                        matrix res(a.GetRows(), a.GetCols(), c);

                        hipFree(c);
                        hipFree(a_);

                        return res;
}
*/
